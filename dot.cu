
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <iostream>

#define WARPSIZE 32
#define TILESIZE 16
#define ELEMS_TILE 256
#define TILEDIM_BLOCK 2 //1blockあたり、16*16の小行列タイルを2*2個生成する
#define TILES_BLOCK 4
using namespace std;
using namespace nvcuda;

//128スレッド4warpで起動されることを想定。2*2のタイルを1blockで計算
//タイルできれいに分割できない行列は未対応
__global__
void dot_TensorCore(float *a, float *b, float *c, int32_t m, int32_t n, int32_t k) {

	//a,b,cでは、小行列の要素が16個ごとにしか連続していない
	//shared memoryには、a,b,cから切り出した部分小行列の各要素が連続して並んでいる状況にする
	__shared__ __half a_half[ELEMS_TILE*TILES_BLOCK] __align__(32);
	__shared__ __half b_half[ELEMS_TILE*TILES_BLOCK] __align__(32);
	__shared__ __half c_half[ELEMS_TILE*TILES_BLOCK] __align__(32);


	int32_t lid = threadIdx.x % WARPSIZE; //warp内の識別id
	int32_t lid_hex = lid % 16;
	int32_t hexid = lid / 16;
	int32_t wid = threadIdx.x / WARPSIZE;
	int32_t tileIdx_x = blockIdx.x * TILEDIM_BLOCK + wid % 2; // 自スレッドがcのx軸方向何枚目のタイル生成担当か
	int32_t tileIdx_y = blockIdx.y * TILEDIM_BLOCK + wid / 2; // 自スレッドがcのy軸以下略

	wmma::fragment<wmma::matrix_a, TILESIZE, TILESIZE, TILESIZE, __half, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, TILESIZE, TILESIZE, TILESIZE, __half, wmma::row_major> b_frag;
	wmma::fragment<wmma::accumulator, TILESIZE, TILESIZE, TILESIZE, __half> c_frag;

	wmma::fill_fragment(c_frag, __float2half(0.f));
	//16*16*16で分割した時にはみ出た部分も余分に計算するために、iの終点を工夫

	if ((tileIdx_y * TILESIZE <= m) && (tileIdx_x * TILESIZE <= n)) { //そもそもcの完全に外に行っているタイルは計算しない

		if(((tileIdx_y + 1) * TILESIZE > m) && ((tileIdx_x + 1) * TILESIZE > n)) { //担当するcのタイルが下にも右にもcからはみ出しているとき

			for (int32_t i=0; i < (k-1) / TILESIZE + 1; i++) {
				if ((i+1) * TILESIZE <= k) { //処理しようとしているタイルがa,bの中に(k方向には)収まっているとき

					int32_t a_offsetbase = tileIdx_y * TILESIZE * k + i * TILESIZE; //a,bの中でのタイルの先頭要素のidx
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if (tileIdx_y * TILESIZE + 2*j + hexid < m) { //自スレッドが処理中の行が、まだaの中に収まってる場合は、aからデータをload
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
						}
						else { //aに収まってない場合は0埋め
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						a_offsetbase += 2 * k; //2行下に移動
					}

					int32_t b_offsetbase = i * TILESIZE * n + tileIdx_x * TILESIZE;
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if (tileIdx_x * TILESIZE + lid_hex < n) { //自スレッドが処理中の列が、まだbの中に収まってる場合は、bからデータをload
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
						}
						else { //bに収まってない場合は0埋め
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						b_offsetbase += 2 * n; //2行下に移動
					}
				}
				else { //ループの最後でa,bから(k方向に)はみ出してしまった時
					int32_t a_offsetbase = tileIdx_y * TILESIZE * k + i * TILESIZE; //a,bの中でのタイルの先頭要素のidx
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if ((tileIdx_y * TILESIZE + 2*j + hexid < m) && (i * TILESIZE + lid_hex < k)) { //自スレッドが処理中の要素が、行方向にも列方向にもまだaの中に収まってる場合は、aからデータをload
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
						}
						else { //aに収まってない場合は0埋め
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						a_offsetbase += 2 * k; //2行下に移動
					}

					int32_t b_offsetbase = i * TILESIZE * n + tileIdx_x * TILESIZE;
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if ((tileIdx_x * TILESIZE + lid_hex < n) && (i * TILESIZE + 2*j + hexid < k)) { //自スレッドが処理中の要素が、行方向にも列方向にもまだbの中に収まってる場合は、bからデータをload
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
						}
						else { //bに収まってない場合は0埋め
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						b_offsetbase += 2 * n; //2行下に移動
					}
				}
				wmma::load_matrix_sync(a_frag, &a_half[wid*ELEMS_TILE], 16);
				wmma::load_matrix_sync(b_frag, &b_half[wid*ELEMS_TILE], 16);
				wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
			}
			wmma::store_matrix_sync(&c_half[wid*ELEMS_TILE], c_frag, 16, wmma::mem_row_major);

			int32_t c_offsetbase = tileIdx_y * TILESIZE * n + tileIdx_x * TILESIZE;
			for (int32_t j=0; j < TILESIZE / 2; j++) {
				if ((tileIdx_y * TILESIZE + 2*j + hexid < m) && (tileIdx_x * TILESIZE + lid_hex < n)) { //自スレッドが処理中の要素が、行方向にも列方向にもまだcの中に収まってる場合だけstore
					c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid + j*32]);
				}
				c_offsetbase += 2 * n; //2行下に移動
			}
		}
		else if ((tileIdx_y + 1) * TILESIZE > m) { //担当するcのタイルが下にだけはみ出している
			for (int32_t i=0; i < (k-1) / TILESIZE + 1; i++) {
				if ((i+1) * TILESIZE <= k) { //処理しようとしているタイルがa,bの中に(k方向には)収まっているとき

					int32_t a_offsetbase = tileIdx_y * TILESIZE * k + i * TILESIZE; //a,bの中でのタイルの先頭要素のidx
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if (tileIdx_y * TILESIZE + 2*j + hexid < m) { //自スレッドが処理中の行が、まだaの中に収まってる場合は、aからデータをload
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
						}
						else { //aに収まってない場合は0埋め
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						a_offsetbase += 2 * k; //2行下に移動
					}

					int32_t b_offsetbase = i * TILESIZE * n + tileIdx_x * TILESIZE;
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
						b_offsetbase += 2 * n; //2行下に移動
					}
				}
				else { //ループの最後でa,bから(k方向に)はみ出してしまった時
					int32_t a_offsetbase = tileIdx_y * TILESIZE * k + i * TILESIZE; //a,bの中でのタイルの先頭要素のidx
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if ((tileIdx_y * TILESIZE + 2*j + hexid < m) && (i * TILESIZE + lid_hex < k)) { //自スレッドが処理中の要素が、行方向にも列方向にもまだaの中に収まってる場合は、aからデータをload
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
						}
						else { //aに収まってない場合は0埋め
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						a_offsetbase += 2 * k; //2行下に移動
					}

					int32_t b_offsetbase = i * TILESIZE * n + tileIdx_x * TILESIZE;
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if (i * TILESIZE + 2*j + hexid < k) { //自スレッドが処理中の要素が列方向にまだbの中に収まってる場合は、bからデータをload
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
						}
						else { //bに収まってない場合は0埋め
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						b_offsetbase += 2 * n; //2行下に移動
					}
				}
				wmma::load_matrix_sync(a_frag, &a_half[wid*ELEMS_TILE], 16);
				wmma::load_matrix_sync(b_frag, &b_half[wid*ELEMS_TILE], 16);
				wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
			}
			wmma::store_matrix_sync(&c_half[wid*ELEMS_TILE], c_frag, 16, wmma::mem_row_major);

			int32_t c_offsetbase = tileIdx_y * TILESIZE * n + tileIdx_x * TILESIZE;
			for (int32_t j=0; j < TILESIZE / 2; j++) {
				if (tileIdx_y * TILESIZE + 2*j + hexid < m) { //自スレッドが処理中の要素が、列方向にまだcの中に収まってる場合だけstore
					c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid + j*32]);
				}
				c_offsetbase += 2 * n; //2行下に移動
			}
		}
		else if ((tileIdx_x + 1) * TILESIZE > n) { //右にだけはみ出している
			for (int32_t i=0; i < (k-1) / TILESIZE + 1; i++) {
				if ((i+1) * TILESIZE <= k) { //処理しようとしているタイルがa,bの中に(k方向には)収まっているとき

					int32_t a_offsetbase = tileIdx_y * TILESIZE * k + i * TILESIZE; //a,bの中でのタイルの先頭要素のidx
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
						a_offsetbase += 2 * k; //2行下に移動
					}

					int32_t b_offsetbase = i * TILESIZE * n + tileIdx_x * TILESIZE;
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if (tileIdx_x * TILESIZE + lid_hex < n) { //自スレッドが処理中の列が、まだbの中に収まってる場合は、bからデータをload
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
						}
						else { //bに収まってない場合は0埋め
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						b_offsetbase += 2 * n; //2行下に移動
					}
				}
				else { //ループの最後でa,bから(k方向に)はみ出してしまった時
					int32_t a_offsetbase = tileIdx_y * TILESIZE * k + i * TILESIZE; //a,bの中でのタイルの先頭要素のidx
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if (i * TILESIZE + lid_hex < k) { //自スレッドが処理中の要素が、行方向にまだaの中に収まってる場合は、aからデータをload
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
						}
						else { //aに収まってない場合は0埋め
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						a_offsetbase += 2 * k; //2行下に移動
					}

					int32_t b_offsetbase = i * TILESIZE * n + tileIdx_x * TILESIZE;
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if ((tileIdx_x * TILESIZE + lid_hex < n) && (i * TILESIZE + 2*j + hexid < k)) { //自スレッドが処理中の要素が、行方向にも列方向にもまだbの中に収まってる場合は、bからデータをload
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
						}
						else { //bに収まってない場合は0埋め
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						b_offsetbase += 2 * n; //2行下に移動
					}
				}
				wmma::load_matrix_sync(a_frag, &a_half[wid*ELEMS_TILE], 16);
				wmma::load_matrix_sync(b_frag, &b_half[wid*ELEMS_TILE], 16);
				wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
			}
			wmma::store_matrix_sync(&c_half[wid*ELEMS_TILE], c_frag, 16, wmma::mem_row_major);

			int32_t c_offsetbase = tileIdx_y * TILESIZE * n + tileIdx_x * TILESIZE;
			for (int32_t j=0; j < TILESIZE / 2; j++) {
				if (tileIdx_x * TILESIZE + lid_hex < n) { //自スレッドが処理中の要素が、行方向にも列方向にもまだcの中に収まってる場合だけstore
					c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid + j*32]);
				}
				c_offsetbase += 2 * n; //2行下に移動
			}
		}
		else { //はみ出してない
			for (int32_t i=0; i < (k-1) / TILESIZE + 1; i++) {
				if ((i+1) * TILESIZE <= k) { //処理しようとしているタイルがa,bの中に収まっているとき
					//a,bの中でのタイルの先頭要素のidx
					int32_t a_offsetbase = tileIdx_y * TILESIZE * k + i * TILESIZE;
					//16*16*16でやろうとしてるので、tidが0~15の担当要素、16~31の担当要素は隔たりがある
					//1回で小行列の2行分をa_halfに。
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
						a_offsetbase += 2 * k; //2行下に移動
					}

					int32_t b_offsetbase = i * TILESIZE * n + tileIdx_x * TILESIZE;
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
						b_offsetbase += 2 * n; //2行下に移動
					}
				}
				else { //このループで最後に16*16のタイル分割時にa,bからはみ出してしまった時
					int32_t a_offsetbase = tileIdx_y * TILESIZE * k + i * TILESIZE;
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if (i * TILESIZE + lid_hex < k) { //自スレッドが処理中の要素が、行方向にまだaの中に収まってる場合は、aからデータをload
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(a[a_offsetbase + hexid*k+lid_hex]);
						}
						else { //aに収まってない場合は0埋め
							a_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						a_offsetbase += 2 * k; //2行下に移動
					}

					int32_t b_offsetbase = i * TILESIZE * n + tileIdx_x * TILESIZE;
					for (int32_t j=0; j < TILESIZE / 2; j++) {
						if (i * TILESIZE + 2*j + hexid < k) { //自スレッドが処理中の要素が、行方向にも列方向にもまだbの中に収まってる場合は、bからデータをload
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(b[b_offsetbase + hexid*n+lid_hex]);
						}
						else { //bに収まってない場合は0埋め
							b_half[wid*ELEMS_TILE + lid + j*32] = __float2half(0.);
						}
						b_offsetbase += 2 * n; //2行下に移動
					}
				}
				wmma::load_matrix_sync(a_frag, &a_half[wid*ELEMS_TILE], 16);
				wmma::load_matrix_sync(b_frag, &b_half[wid*ELEMS_TILE], 16);
				wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
			}
			wmma::store_matrix_sync(&c_half[wid*ELEMS_TILE], c_frag, 16, wmma::mem_row_major);

			int32_t c_offsetbase = tileIdx_y * TILESIZE * n + tileIdx_x * TILESIZE;
			for (int32_t j=0; j < TILESIZE / 2; j++) {
				c[c_offsetbase + hexid*n+lid_hex] = __half2float(c_half[wid*ELEMS_TILE + lid + j*32]);
				c_offsetbase += 2 * n; //2行下に移動
			}
		}
	}
}

int main() {
	int32_t m = 33;
	int32_t k = 36;
	int32_t n = 34;
	int32_t a_matsize = m * k;
	int32_t b_matsize = k * n;
	int32_t c_matsize = m * n;
	float *a, *b, *c;
	float *a_dev, *b_dev, *c_dev;
	a = (float*)malloc(sizeof(float) * a_matsize);
	b = (float*)malloc(sizeof(float) * b_matsize);
	c = (float*)malloc(sizeof(float) * c_matsize);

	hipMalloc((void**)&a_dev, sizeof(float) * a_matsize);
	hipMalloc((void**)&b_dev, sizeof(float) * b_matsize);
	hipMalloc((void**)&c_dev, sizeof(float) * c_matsize);
	for (int32_t i=0; i < a_matsize; i++) {
		a[i] = (float)i;
	}
	for (int32_t i=0; i < b_matsize; i++) {
		b[i] = 0.;
	}
	for (int32_t i=0; i < c_matsize; i++) {
		c[i] = 0.;
	}
	for (int32_t i=0; i < n; i++) {
		b[i] = (float)i;
	}
	hipMemcpy(a_dev, a, sizeof(float)*a_matsize, hipMemcpyHostToDevice);
	hipMemcpy(b_dev, b, sizeof(float)*b_matsize, hipMemcpyHostToDevice);
	dim3 grid;
	grid.x = 2;
	grid.y = 2;
	dot_TensorCore<<<grid, 128>>>(a_dev,  b_dev, c_dev, m, n, k);
	hipMemcpy(c, c_dev, sizeof(float)*c_matsize, hipMemcpyDeviceToHost);
	for (int32_t i=0; i < m; i++) {
		for (int32_t j=0; j < n; j++) {
			cout << c[i*n+j] << " ";
		}
		cout << endl;
	}
	return 0;
}
